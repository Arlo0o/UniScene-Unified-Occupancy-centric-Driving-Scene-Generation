#include <hip/hip_runtime.h>
#include <math.h>

#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define DIVUP(m, n) ((m + n - 1) / n)

__global__ void dda3d_kernel(
    const float* rays, const bool* grid, bool* intersections,
    float Xmin, float Ymin, float Zmin, float vx, float vy, float vz,
    int W, int H, int D, int num_rays) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_rays) return;
 
    float dx = rays[i * 3 + 0];
    float dy = rays[i * 3 + 1];
    float dz = rays[i * 3 + 2];

 
    int x = (0 - Xmin) / vx;
    int y = (0 - Ymin) / vy;
    int z = (0 - Zmin) / vz;

    int stepX = (dx > 0) ? 1 : -1;
    int stepY = (dy > 0) ? 1 : -1;
    int stepZ = (dz > 0) ? 1 : -1;

    float tDeltaX = (dx != 0) ? abs(vx / dx) : INFINITY;
    float tDeltaY = (dy != 0) ? abs(vy / dy) : INFINITY;
    float tDeltaZ = (dz != 0) ? abs(vz / dz) : INFINITY;

    float tMaxX = (dx > 0) ? (Xmin + (x + 1) * vx) / dx : (Xmin + x * vx) / dx;
    float tMaxY = (dy > 0) ? (Ymin + (y + 1) * vy) / dy : (Ymin + y * vy) / dy;
    float tMaxZ = (dz > 0) ? (Zmin + (z + 1) * vz) / dz : (Zmin + z * vz) / dz;

 
    while ((0 <= x && x < W) && (0 <= y && y < H) && (0 <= z && z < D)) {
     
        int grid_index = z + y * D + x * H * D;

      
        if (grid[grid_index]) {
            intersections[i] = true;
            return;
        }

    
        if (tMaxX < tMaxY) {
            if (tMaxX < tMaxZ) {
                x += stepX;
                tMaxX += tDeltaX;
            } else {
                z += stepZ;
                tMaxZ += tDeltaZ;
            }
        } else {
            if (tMaxY < tMaxZ) {
                y += stepY;
                tMaxY += tDeltaY;
            } else {
                z += stepZ;
                tMaxZ += tDeltaZ;
            }
        }
    }
    intersections[i] = false;   
}


__global__ void raycast_kernel(
    const float* rays, const bool* grid, bool* intersections, int* hits,
    float Xmin, float Ymin, float Zmin, float vx, float vy, float vz,
    int W, int H, int D, int num_rays) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_rays) return;

 
    float dx = rays[i * 3 + 0];
    float dy = rays[i * 3 + 1];
    float dz = rays[i * 3 + 2];
 
    int x = (0 - Xmin) / vx;
    int y = (0 - Ymin) / vy;
    int z = (0 - Zmin) / vz;

    int stepX = (dx > 0) ? 1 : -1;
    int stepY = (dy > 0) ? 1 : -1;
    int stepZ = (dz > 0) ? 1 : -1;

    float tDeltaX = (dx != 0) ? abs(vx / dx) : INFINITY;
    float tDeltaY = (dy != 0) ? abs(vy / dy) : INFINITY;
    float tDeltaZ = (dz != 0) ? abs(vz / dz) : INFINITY;

    float tMaxX = (dx > 0) ? (Xmin + (x + 1) * vx) / dx : (Xmin + x * vx) / dx;
    float tMaxY = (dy > 0) ? (Ymin + (y + 1) * vy) / dy : (Ymin + y * vy) / dy;
    float tMaxZ = (dz > 0) ? (Zmin + (z + 1) * vz) / dz : (Zmin + z * vz) / dz;

  
    while ((0 <= x && x < W) && (0 <= y && y < H) && (0 <= z && z < D)) {
      
        int grid_index = z + y * D + x * H * D;

     
        if (grid[grid_index]) {
            intersections[i] = true;
            hits[i] = x;
            hits[i + 1] = y;
            hits[i + 2] = z;
            return;
        }

        
        if (tMaxX < tMaxY) {
            if (tMaxX < tMaxZ) {
                x += stepX;
                tMaxX += tDeltaX;
            } else {
                z += stepZ;
                tMaxZ += tDeltaZ;
            }
        } else {
            if (tMaxY < tMaxZ) {
                y += stepY;
                tMaxY += tDeltaY;
            } else {
                z += stepZ;
                tMaxZ += tDeltaZ;
            }
        }
    }
    intersections[i] = false;   
}

void dda3d_launcher(const float* rays, const bool* grid, bool* intersections,
    float Xmin, float Ymin, float Zmin, float vx, float vy, float vz,
    int W, int H, int D, int num_rays){
    dim3 blockSize(DIVUP(num_rays, THREADS_PER_BLOCK));
    dim3 threadSize(THREADS_PER_BLOCK);
    dda3d_kernel<<<blockSize, threadSize>>>(rays, grid, intersections, Xmin, Ymin, Zmin, vx, vy, vz, W, H, D, num_rays);
}

void raycast_launcher(const float* rays, const bool* grid, bool* intersections, int* hits,
    float Xmin, float Ymin, float Zmin, float vx, float vy, float vz,
    int W, int H, int D, int num_rays){
    dim3 blockSize(DIVUP(num_rays, THREADS_PER_BLOCK));
    dim3 threadSize(THREADS_PER_BLOCK);
    raycast_kernel<<<blockSize, threadSize>>>(rays, grid, intersections, hits, Xmin, Ymin, Zmin, vx, vy, vz, W, H, D, num_rays);
}